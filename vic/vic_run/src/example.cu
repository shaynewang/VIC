#include "hip/hip_runtime.h"
#include <iostream>

extern "C" {
#include "vic_run.h"
}


__global__
void some_kernel(void) {
    int x = 1;
    x += 1;
}



using namespace std;


extern "C" 
void example_print() {
    cout << "Cuda is linked <---------" << endl;
    some_kernel<<<1, 1>>>();
}


extern "C" 
void init_layer_data(layer_data_struct* obj, layer_data_struct* val, int size) {
    layer_data_struct* d_obj;
    layer_data_struct* d_val;

    hipMalloc((void**) &d_obj, size);
    hipMalloc((void**) &d_val, size);

    hipMemcpy(d_obj, obj, size,  hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, size, hipMemcpyHostToDevice);
}

